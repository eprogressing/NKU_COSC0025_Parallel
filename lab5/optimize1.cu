#include <cstring>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>
#include <iomanip>
#include <cmath>
#include <vector>

#include <hip/hip_runtime.h>

// CUDA 错误检查宏
#define CUDA_CHECK(err) { \
    hipError_t e = err; \
    if (e != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(e) << " in " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// =================================================================
// HELPER & PRECOMPUTATION
// =================================================================

// Host-side power function for precomputation
long long power_host(long long base, long long exp, long long mod) {
    long long res = 1;
    base %= mod;
    while (exp > 0) {
        if (exp % 2 == 1) res = (res * base) % mod;
        base = (base * base) % mod;
        exp /= 2;
    }
    return res;
}

// Precomputes all necessary twiddle factors on the host
void precompute_twiddle_factors(std::vector<int>& wn, int n, int p, bool is_inverse) {
    int g = 3; // Primitive root
    if (n <= 1) return;
    // FIX 1: The total number of twiddle factors needed for all stages is n-1.
    // The sum of m/2 for m = 2, 4, ..., n is 1 + 2 + 4 + ... + n/2 = n-1.
    wn.resize(n - 1);
    
    int offset = 0;
    for (int m = 2; m <= n; m <<= 1) {
        long long wm_base = power_host(g, (p - 1) / m, p);
        if (is_inverse) {
            wm_base = power_host(wm_base, p - 2, p);
        }

        long long w = 1;
        for (int j = 0; j < m / 2; j++) {
            wn[offset + j] = w;
            w = (w * wm_base) % p;
        }
        offset += m / 2;
    }
}


// =================================================================
// GPU KERNELS and DEVICE FUNCTIONS
// =================================================================

__device__ long long power_gpu(long long base, long long exp, long long mod) {
    long long res = 1;
    base %= mod;
    while (exp > 0) {
        if (exp % 2 == 1) res = (res * base) % mod;
        base = (base * base) % mod;
        exp /= 2;
    }
    return res;
}


__global__ void bit_reverse_kernel(int* a, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return; // Boundary check

    int logn = 0;
    if (n > 1) logn = __log2f(n);
    
    int rev_i = 0;
    for (int j = 0; j < logn; j++) {
        if ((i >> j) & 1) {
            rev_i |= 1 << (logn - 1 - j);
        }
    }

    // Only one of the threads in a pair performs the swap.
    if (i < rev_i) {
        int temp = a[i];
        a[i] = a[rev_i];
        a[rev_i] = temp;
    }
}

// OPTIMIZED KERNEL: Reads twiddle factors from a precomputed table
__global__ void ntt_stage_kernel_optimized(int* a, const int* wn, int n, int m, int stage_offset, int p) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n / 2) return; 

    int j = tid % (m / 2);
    int k = (tid / (m / 2)) * m;

    int idx1 = k + j;
    int idx2 = idx1 + m / 2;

    // Read precomputed twiddle factor instead of calculating it
    long long w = wn[stage_offset + j];
    // FIX 2: Use the passed-in modulus 'p' instead of a hardcoded value.
    long long t = (w * a[idx2]) % p; 
    long long u = a[idx1];
    
    a[idx1] = (u + t) % p;
    a[idx2] = (u - t + p) % p;
}

__global__ void pointwise_mult_kernel(int* a, int* b, int* ab, int n, int p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        ab[i] = (1LL * a[i] * b[i]) % p;
    }
}

__global__ void normalize_kernel(int* a, int n, int p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        long long n_inv = power_gpu(n, p - 2, p);
        a[i] = (1LL * a[i] * n_inv) % p;
    }
}

// =================================================================
// GPU WRAPPER FUNCTION (The new poly_multiply)
// =================================================================
void ntt_gpu(int* d_a, int* d_wn, int n, int p, bool is_inverse) {
    int threadsPerBlock = 256;
    
    int blocks_full = (n + threadsPerBlock - 1) / threadsPerBlock;
    bit_reverse_kernel<<<blocks_full, threadsPerBlock>>>(d_a, n);
    
    // Butterfly stages - launch kernels without explicit sync
    int stage_offset = 0;
    for (int m = 2; m <= n; m <<= 1) {
        int stage_blocks = (n / 2 + threadsPerBlock - 1) / threadsPerBlock;
        // FIX 3: Pass 'p' to the kernel.
        ntt_stage_kernel_optimized<<<stage_blocks, threadsPerBlock>>>(d_a, d_wn, n, m, stage_offset, p);
        stage_offset += m / 2;
    }

    if (is_inverse) {
        normalize_kernel<<<blocks_full, threadsPerBlock>>>(d_a, n, p);
    }
}


void poly_multiply(int* h_a, int* h_b, int* h_ab, int n, int p) {
    int m = 1;
    while (m < 2 * n) { m <<= 1; }
    
    std::vector<int> a_padded(m, 0);
    std::vector<int> b_padded(m, 0);
    for(int i = 0; i < n; i++) {
        a_padded[i] = h_a[i];
        b_padded[i] = h_b[i];
    }

    // 1. Precompute twiddle factors on host
    std::vector<int> h_wn_fwd, h_wn_inv;
    precompute_twiddle_factors(h_wn_fwd, m, p, false);
    precompute_twiddle_factors(h_wn_inv, m, p, true);

    int *d_a, *d_b, *d_wn_fwd, *d_wn_inv;
    size_t size_m = m * sizeof(int);
    // FIX 4: The correct size for the twiddle factor array is (m-1).
    size_t size_wn = (m > 1) ? (m - 1) * sizeof(int) : 0;

    // 2. Allocate memory on device (including for twiddles)
    CUDA_CHECK(hipMalloc((void**)&d_a, size_m));
    CUDA_CHECK(hipMalloc((void**)&d_b, size_m));
    if (size_wn > 0) {
        CUDA_CHECK(hipMalloc((void**)&d_wn_fwd, size_wn));
        CUDA_CHECK(hipMalloc((void**)&d_wn_inv, size_wn));
    }

    // 3. Copy data to device
    CUDA_CHECK(hipMemcpy(d_a, a_padded.data(), size_m, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b_padded.data(), size_m, hipMemcpyHostToDevice));
    if (size_wn > 0) {
        CUDA_CHECK(hipMemcpy(d_wn_fwd, h_wn_fwd.data(), size_wn, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_wn_inv, h_wn_inv.data(), size_wn, hipMemcpyHostToDevice));
    }

    // Forward NTT
    ntt_gpu(d_a, d_wn_fwd, m, p, false);
    ntt_gpu(d_b, d_wn_fwd, m, p, false);

    // Pointwise Multiplication
    int threads = 256;
    int blocks = (m + threads - 1) / threads;
    pointwise_mult_kernel<<<blocks, threads>>>(d_a, d_b, d_a, m, p);
    
    // Inverse NTT
    ntt_gpu(d_a, d_wn_inv, m, p, true);
    
    // All kernels are queued, now wait for all to finish
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    std::vector<int> ab_padded(m);
    CUDA_CHECK(hipMemcpy(ab_padded.data(), d_a, size_m, hipMemcpyDeviceToHost));
    
    for(int i = 0; i < 2 * n - 1; i++) {
        h_ab[i] = ab_padded[i];
    }

    // 4. Free all device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    if (size_wn > 0) {
        CUDA_CHECK(hipFree(d_wn_fwd));
        CUDA_CHECK(hipFree(d_wn_inv));
    }
}

// =================================================================
// USER'S ORIGINAL FRAMEWORK (UNCHANGED, except for file paths)
// =================================================================
void fRead(int *a, int *b, int *n, int *p, int input_id){
    // Use relative path
    std::string str1 = "nttdata/";
    std::string str2 = std::to_string(input_id);
    std::string strin = str1 + str2 + ".in";

    std::ifstream fin(strin);
    if (!fin.is_open()) {
        std::cerr << "Error opening input file: " << strin << std::endl;
        *n = 0; // Signal failure
        return;
    }
    fin>>*n>>*p;
    for (int i = 0; i < *n; i++){
        fin>>a[i];
    }
    for (int i = 0; i < *n; i++){   
        fin>>b[i];
    }
    fin.close();
}

void fCheck(int *ab, int n, int input_id){
    // Use relative path
    std::string str1 = "nttdata/";
    std::string str2 = std::to_string(input_id);
    std::string strout = str1 + str2 + ".out";

    std::ifstream fin(strout);
    if (!fin.is_open()) {
        std::cerr << "Error opening output check file: " << strout << std::endl;
        return;
    }
    for (int i = 0; i < n * 2 - 1; i++){
        int x;
        fin>>x;
        if (!fin) {
             std::cout << "Error reading from check file or file ended prematurely." << std::endl;
             break;
        }
        if(x != ab[i]){
            std::cout<<"多项式乘法结果错误 at index " << i << ". Expected " << x << ", got " << ab[i] << std::endl;
            fin.close();
            return;
        }
    }
    std::cout<<"多项式乘法结果正确"<<std::endl;
    fin.close();
}

void fWrite(int *ab, int n, int input_id){
    // Use relative path
    std::string str1 = "files/";
    std::string str2 = std::to_string(input_id);
    std::string strout = str1 + str2 + ".out";
    
    std::ofstream fout(strout);
    for (int i = 0; i < n * 2 - 1; i++){
        fout<<ab[i]<<'\n';
    }
    fout.close();
}

int a[300000], b[300000], ab[300000];

int main(int argc, char *argv[])
{
    // Make sure 'files' and 'nttdata' directories exist
    // For test '1.in', n=131072, p=7340033
    int test_begin = 0;
    int test_end = 3; 

    for(int i = test_begin; i <= test_end; ++i){
        long double ans = 0;
        int n_, p_;
        fRead(a, b, &n_, &p_, i);
        if (n_ == 0) continue; // Skip if file read failed

        memset(ab, 0, sizeof(ab));
        auto Start = std::chrono::high_resolution_clock::now();
        
        poly_multiply(a, b, ab, n_, p_);
        
        auto End = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double,std::milli> elapsed = End - Start;
        ans += elapsed.count();
        
        fCheck(ab, n_, i);
        std::cout<<"Latency for n = "<<n_<<", p = "<<p_<<" : "<<ans<<" (ms) "<<std::endl;
        
        fWrite(ab, n_, i);
    }
    return 0;
}

